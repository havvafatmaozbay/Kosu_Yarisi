#include "hip/hip_runtime.h"
#include ""

#include "cpu_gpu_mem.h"
#include "kernel_gpu_racer.cuh"

__global__ void gpu_racer(int* gpu_numbers, const int nc)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < nc) {
		gpu_numbers[id] *= 1;
	}

}

void cpu_gpu_execute(Racer* racer)
{
	int number_count = racer->nc;

	//int blockDim = 64;
	//int gridDim = (number_count + blockDim - 1) / blockDim;

	//execute
	gpu_racer << <1, 100 >> > ((int*)racer->racerID_gpu, number_count);
	gpu_racer << <1, 100 >> > ((int*)racer->racerSpeed_gpu, number_count);
	gpu_racer << <1, 100 >> > ((int*)racer->racerLocation_gpu, number_count);
}